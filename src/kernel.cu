#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <ctime>
#include <iomanip>
#include <chrono>

#include "traders.cuh"
#include "cudamacro.h"
#include "cudahelper.h"

#define timer std::chrono::high_resolution_clock

using namespace std;


map<string, string> read_config_file(string config_filename, string delimiter = "=")
{
    std::ifstream config_file;
    config_file.open(config_filename);
    map<string, string> config;

    if (!config_file.is_open()) {
        std::cout << "Could not open file '" << config_filename << "'" << std::endl;
        return config;
    } else {
        int row = 0;
        std::string line = "";
        std::string key = "";

        std::cout << "Launch configuration:" << std::endl;

        while (getline(config_file, line)) {
            if (line[0] == '#' || line == "") continue;
            int delimiter_position = line.find(delimiter);

            for (int idx = 0; idx < delimiter_position; idx++) {
                if (line[idx] != ' ') key += line[idx];
            }

            std::string value = line.substr(delimiter_position + 1, line.length() - 1);
            config[key] = value;
            std::cout << '\t' << key << ": ";
            std::cout << value << std::endl;
            row++;
            key = "";
        }
        config_file.close();
        return config;
    }
}


int main(int argc, char** argv) {

    std::ofstream file;
    signed char *d_black_tiles, *d_white_tiles, *d_black_plus_white;
    float *random_values, *d_probabilities;
    hiprandGenerator_t rng;
    // The global market represents the sum over the strategies of each
    // agent. Agents will choose a strategy contrary to the sign of the
    // global market.
    int device_id = 0;
    string config_filename = (argc == 1) ? "ising2d.conf" : argv[1];
    map<string, string> config = read_config_file(config_filename);

    //TODO ternary operator to replace with default arg if not passed.
    const long long grid_height = std::stoll(config["grid_height"]);
    const long long grid_width = std::stoll(config["grid_width"]);
    unsigned int total_updates = std::stoul(config["total_updates"]);
    unsigned int seed = std::stoul(config["seed"]);
    float alpha = std::stof(config["alpha"]);
    float j = std::stof(config["j"]);
    float beta = std::stof(config["beta"]);
    // the rng offset can be used to return the random number generator to a specific
    // state of a simulation. It is equal to the total number of random numbers
    // generated. Meaning the following equation holds for this specific case:
    // rng_offset = (total_updates + 1) * grid_width * grid_height
    // (+ 1 because of the random numbers created for the initilisation)
    unsigned long long rng_offset = (config["rng_offset"] != "") ? stoull(config["rng_offset"]) : 0;
    float reduced_alpha = -2.0f * beta * alpha;
    float reduced_j = -2.0f * beta * j;

    // finds and sets the specified cuda device
    gpuDeviceInit(device_id);
    // Finds and prints the devices name and computing power
    hipDeviceProp_t deviceProp;
    deviceProp.major = 0;
    deviceProp.minor = 0;
    CHECK_CUDA(hipGetDeviceProperties(&deviceProp, device_id));
    printf("CUDA device [%s] has %d Multi-Processors, Compute %d.%d\n",
        deviceProp.name, deviceProp.multiProcessorCount, deviceProp.major, deviceProp.minor);


    // Set up cuRAND generator
    CHECK_CURAND(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
    CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(rng, seed));
    CHECK_CURAND(hiprandSetGeneratorOffset(rng, rng_offset));

    // allocate memory for the arrays
    CHECK_CUDA(hipMalloc(&d_white_tiles, grid_height * grid_width / 2 * sizeof(*d_white_tiles)));
    CHECK_CUDA(hipMalloc(&d_black_tiles, grid_height * grid_width / 2 * sizeof(*d_black_tiles)));
    CHECK_CUDA(hipMalloc(&d_black_plus_white, grid_height * grid_width / 2 * sizeof(*d_black_plus_white)));
    CHECK_CUDA(hipMalloc(&random_values, grid_height * grid_width / 2 * sizeof(*random_values)));
    CHECK_CUDA(hipMalloc(&d_probabilities, 10 * sizeof(*random_values)));

    init_traders(d_black_tiles, d_white_tiles, rng, random_values, grid_width, grid_height);
    // Synchronize operations on the GPU with CPU
    CHECK_CUDA(hipDeviceSynchronize());


    file.open("magnetisation.dat");
    timer::time_point start = timer::now();
    for (int iteration = 0; iteration < total_updates; iteration++) {
        float global_market = update(d_black_tiles, d_white_tiles, d_black_plus_white, random_values,
                                     d_probabilities, rng, reduced_alpha, reduced_j, grid_height, grid_width);
        file << global_market << std::endl;
    }
    timer::time_point stop = timer::now();
    file.close();
    file.open("logs/ising.log", std::ios_base::app);
    auto t = std::time(nullptr);
    auto tm = *std::localtime(&t);
    file << std::put_time(&tm, "%d.%m.%Y %H:%M:%S") << std::endl;
    file << grid_width << 'x' << grid_height << std::endl;
    file << "seed: " << seed << std::endl;
    file << "total updates: " << total_updates << std::endl;

    double duration = (double) std::chrono::duration_cast<std::chrono::microseconds>(stop - start).count();
    double spin_updates_per_nanosecond = total_updates * grid_width * grid_height / duration * 1e-3;
    printf("Total computing time: %f\n", duration * 1e-6);
    file << "total computing time: " << std::to_string(duration * 1e-6) << std::endl;
    printf("Updates per nanosecond: %f\n", spin_updates_per_nanosecond);
    file << "updates per nanosecond: " << std::to_string(spin_updates_per_nanosecond) << std::endl;
    file << "-----------------------------------" << std::endl;
    file.close();
    file.open("log");
    file << "updates/ns: " << spin_updates_per_nanosecond << std::endl;
    file.close();
    CHECK_CUDA(hipDeviceSynchronize());
    return 0;
}
